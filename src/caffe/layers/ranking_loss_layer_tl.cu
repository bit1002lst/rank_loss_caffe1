#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void RealRankingLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_sub(
      count,
      bottom[2]->gpu_data(),  // G(a)
      bottom[3]->gpu_data(),  // G(b)
      diff2_.mutable_gpu_data());  // G(a)_i-G(b)_i
//caffe_gpu_axpby(
//      count,
//      Dtype(0.0125),
//      diff2_.cpu_data(),
//      Dtype(0),
//      rescaled_.mutable_cpu_data());

  Dtype margin = this->layer_param_.real_ranking_loss_param().margin();
    //bool legacy_version =
  //    this->layer_param_.real_ranking_loss_param().legacy_version();
  Dtype loss(0.0);
  //LOG(INFO) << "HI! ";
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (diff2_.cpu_data()[i] > 0) {  // if first is larger (a-b)
        loss += std::max(margin - diff_.cpu_data()[i], Dtype(0.0));
        //dist_mine_.mutable_gpu_data()[i]=diff_.cpu_data()[i];
        //LOG(INFO) << "BYE! ";
    }
    else {
        //LOG(INFO) << margin << " " << -diff_.cpu_data()[i];
        loss += std::max(margin + diff_.cpu_data()[i], Dtype(0.0));
        //dist_mine_.mutable_gpu_data()[i]= -diff_.cpu_data()[i];
        //LOG(INFO) << "BYE! ";
    }
//LOG(INFO) <<  "G(a)-G(b): " << diff2_.cpu_data()[i] << "; a-b: " << diff_.cpu_data()[i];
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num());
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const Dtype alpha,
    const Dtype* y, const Dtype* diff, Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_mine
    Dtype mdist(0.0);
    if (y[n] > 0.0) {  // first is larger
        mdist = (margin - diff[n]);
        if (mdist > 0.0) {
            bottom_diff[i] = -alpha;
        } else {
            bottom_diff[i] = 0;
        }
    }
    else {
        mdist = (margin + diff[n]);
        if (mdist > 0.0) {
            bottom_diff[i] = alpha;
        }
        else {
        bottom_diff[i] = 0;
        }
    }
  }
}

template <typename Dtype>
void RealRankingLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const int count = bottom[0]->count();
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {

      const int channels = bottom[0]->channels(); //1
      Dtype margin = this->layer_param_.real_ranking_loss_param().margin();
      //const bool legacy_version =
      //    this->layer_param_.real_ranking_loss_param().legacy_version();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, alpha,
          diff2_.gpu_data(),  // pair similarity 0 or 1
          diff_.gpu_data(),
          bottom[i]->mutable_gpu_diff());

      CUDA_POST_KERNEL_CHECK;
    }
  }
//int i = 0;
//for (int j = 0; j < count; ++j) {
//LOG(INFO) <<  "G(a)-G(b): " << diff2_.cpu_data()[j] << "; a-b: " << diff_.cpu_data()[j] << "; diff: " << bottom[i]->cpu_diff()[j];
//}
}

INSTANTIATE_LAYER_GPU_FUNCS(RealRankingLossLayer);

}  // namespace caffe
